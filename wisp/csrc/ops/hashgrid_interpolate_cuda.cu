#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
 *
 * NVIDIA CORPORATION & AFFILIATES and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION & AFFILIATES is strictly prohibited.
 */

#include <ATen/ATen.h>
#include <c10/cuda/CUDAGuard.h>

namespace wisp {
typedef unsigned int uint;

__device__ int32_t
hash_index(
    const int3 pos,
    const int32_t resolution,
    const int32_t codebook_size
){
    int32_t index = 0;

    constexpr uint32_t primes[3] = { 1u, 2654435761u, 805459861u };

    if (resolution < codebook_size &&
        resolution * resolution < codebook_size &&
        resolution * resolution * resolution < codebook_size) {
        index = pos.x +
                pos.y * resolution +
                pos.z * resolution * resolution;
    } else {
        index = (pos.x * primes[0] ^
                 pos.y * primes[1] ^
                 pos.z * primes[2]) % codebook_size;
    }
    return index;
}

__device__ float
clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__global__ void
hashgrid_interpolate_cuda_kernel(
    const int64_t num_coords,
    const int32_t codebook_size,
    const int64_t feature_dim,
    const int32_t resolution,
    const int32_t lod_idx,
    const int32_t num_lods,
    const float* coords,
    const float* codebook,
    float* feats
){
    uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int64_t stride = blockDim.x*gridDim.x;
    for (int64_t i=tidx; i<num_coords; i+=stride) {

        float3 x = make_float3(clamp(resolution * (coords[i*3+0] * 0.5 + 0.5), 0, resolution-1-1e-5),
                               clamp(resolution * (coords[i*3+1] * 0.5 + 0.5), 0, resolution-1-1e-5),
                               clamp(resolution * (coords[i*3+2] * 0.5 + 0.5), 0, resolution-1-1e-5));
        int3 pos = make_int3(floor(x.x), floor(x.y), floor(x.z));
        float3 x_ = make_float3(x.x - (float) pos.x, x.y - (float) pos.y, x.z - (float) pos.z);
        float3 _x = make_float3(1.0 - x_.x, 1.0 - x_.y, 1.0 - x_.z);

        float c000 = _x.x * _x.y * _x.z;
        float c001 = _x.x * _x.y * x_.z;
        float c010 = _x.x * x_.y * _x.z;
        float c011 = _x.x * x_.y * x_.z;
        float c100 = x_.x * _x.y * _x.z;
        float c101 = x_.x * _x.y * x_.z;
        float c110 = x_.x * x_.y * _x.z;
        float c111 = x_.x * x_.y * x_.z;

        int32_t corner_idx[8];
#       pragma unroll
        for (int j=0; j<8; ++j) {
            int3 corner;
            corner.x = pos.x + ((j & 4) >> 2);
            corner.y = pos.y + ((j & 2) >> 1);
            corner.z = pos.z + ((j & 1) >> 0);
            corner_idx[j] = hash_index(corner, resolution, codebook_size);
        }

        for (uint64_t j=0; j<feature_dim; ++j) {
            float c[8];
            for (int m = 0; m < 8; m++)
            {
                c[m] = codebook[corner_idx[m] * feature_dim + j];
            }

            float feat =
                c[0] * c000 +
                c[1] * c001 +
                c[2] * c010 +
                c[3] * c011 +
                c[4] * c100 +
                c[5] * c101 +
                c[6] * c110 +
                c[7] * c111;
            feats[num_lods*i*feature_dim+feature_dim*lod_idx+j] = feat;


            printf("0: %f \n 1: %f \n 2: %f \n 3: %f \n 4: %f \n 5: %f \n 6: %f \n 7: %f\n",
                   c[0], c[1], c[2], c[3], c[4],
                   c[5], c[6], c[7]);
        }

/*
        // --- Yasasa - Temp debug
        printf("----- FORWARD ------\n");
        printf("0: %d \n 1: %d \n 2: %d \n 3: %d \n 4: %d \n 5: %d \n 6: %d \n 7: %d\n",
               corner_idx[0], corner_idx[1], corner_idx[2], corner_idx[3], corner_idx[4],
               corner_idx[5], corner_idx[6], corner_idx[7]);


        printf("0: %f \n 1: %f \n 2: %f \n 3: %f \n 4: %f \n 5: %f \n 6: %f \n 7: %f\n",
               c000, c001, c010, c011, c100,
               c101, c110, c111);
        printf("Randomteststuff\n");
        // Yasasa - Temp debug
        */
    }
}

void hashgrid_interpolate_cuda_impl(
    int64_t num_coords,
    int32_t codebook_size,
    int64_t feature_dim,
    int32_t resolution,
    int32_t lod_idx,
    int32_t num_lods,
    at::Tensor coords,
    at::Tensor codebook,
    at::Tensor feats){

    int num_threads = 512;

    const at::cuda::OptionalCUDAGuard device_guard(at::device_of(feats));
    auto stream = at::cuda::getCurrentCUDAStream();
    hashgrid_interpolate_cuda_kernel<<<(num_coords + num_threads - 1) / num_threads, num_threads, 0, stream>>>(
        num_coords,
        codebook_size,
        feature_dim,
        resolution,
        lod_idx,
        num_lods,
        coords.data_ptr<float>(),
        codebook.data_ptr<float>(),
        feats.data_ptr<float>()
    );
}
// --- Yasasa - interp gradients

__device__ void calc_grad_wrt_x_(int64_t feat_idx, float resolution, int64_t feature_dim, float3 x_, float3 _x, int32_t corner_idx[8], float gradout, float* grad_x_, float* codebook){
    float c[8];
    int64_t j = feat_idx;
    for(int i = 0; i < 8; i++){
        c[i] = codebook[corner_idx[i]*feature_dim+j];
    }

    gradout *= resolution / 2.f;

    auto x_grad = _x.y * _x.z * (c[4] - c[0])
                + x_.y * _x.z * (c[6] - c[2])
                + _x.y * x_.z * (c[5] - c[1])
                + x_.y * x_.z * (c[7] - c[3]);

    grad_x_[0] = gradout * x_grad;
    //atomicAdd(grad_x_, gradout * x_grad);

    auto y_grad = _x.x * _x.z * (c[2] - c[0]) +
                  _x.x * x_.z * (c[3] - c[1]) +
                  x_.x * _x.z * (c[6] - c[4]) +
                  x_.x * x_.z * (c[7] - c[5]);

    float c00 = c[0]*_x.x + c[4]*x_.x;
    float c01 = c[1]*_x.x + c[5]*x_.x;
    float c10 = c[2]*_x.x + c[6]*x_.x;
    float c11 = c[3]*_x.x + c[7]*x_.x;
   // auto y_grad =  _x.z * (c10 - c00)
    //             + x_.z * (c11 - c01);

    grad_x_[1] = gradout * y_grad;
    //atomicAdd(grad_x_ + 1, gradout * y_grad);


    auto z_grad = _x.x * _x.y * (c[1] - c[0]) +
                  _x.x * x_.y * (c[3] - c[2]) +
                  x_.x * _x.y * (c[5] - c[4]) +
                  x_.x * x_.y * (c[7] - c[6]);

    //float c0 = c00 * _x.y + c10 * x_.y;
   // float c1 = c01 * _x.y + c11 * x_.y;
   // auto z_grad = (c1 - c0);
    grad_x_[2] = gradout * z_grad;
    //atomicAdd(grad_x_ + 2, gradout * z_grad);
    printf("BACKWARD\n");
}

// gradient of interpolated features w.r.t grid points
// --- Yasasa - interp gradients
__global__ void
hashgrid_interpolate_backward_cuda_kernel(
    const int64_t num_coords,
    const int32_t codebook_size,
    const int64_t feature_dim,
    const int32_t resolution,
    const int32_t lod_idx,
    const int32_t num_lods,
    const float* coords,
    const float* grad_output, // N, feature_dim*num_lods
    float* grad_codebook, // codebook_size, feature_dim
    float* codebook,
    float* grad_coords // num_coords * num_features, 3
){
    uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int64_t stride = blockDim.x*gridDim.x;
    for (int64_t i=tidx; i<num_coords; i+=stride) {

        float3 x = make_float3(clamp(resolution * (coords[i*3+0] * 0.5 + 0.5), 0, resolution-1-1e-5),
                               clamp(resolution * (coords[i*3+1] * 0.5 + 0.5), 0, resolution-1-1e-5),
                               clamp(resolution * (coords[i*3+2] * 0.5 + 0.5), 0, resolution-1-1e-5));
        int3 pos = make_int3(floor(x.x), floor(x.y), floor(x.z));
        float3 x_ = make_float3(x.x - (float) pos.x, x.y - (float) pos.y, x.z - (float) pos.z);
        float3 _x = make_float3(1.0 - x_.x, 1.0 - x_.y, 1.0 - x_.z);

        printf("%f %f %f %d %d %d \n", x_.x, x_.y, x_.z, pos.x, pos.y, pos.z);

        float coeffs[8];
        coeffs[0] = _x.x * _x.y * _x.z;
        coeffs[1] = _x.x * _x.y * x_.z;
        coeffs[2] = _x.x * x_.y * _x.z;
        coeffs[3] = _x.x * x_.y * x_.z;
        coeffs[4] = x_.x * _x.y * _x.z;
        coeffs[5] = x_.x * _x.y * x_.z;
        coeffs[6] = x_.x * x_.y * _x.z;
        coeffs[7] = x_.x * x_.y * x_.z;

        int32_t corner_idx[8];
#       pragma unroll
        for (int j=0; j<8; ++j) {
            int3 corner;
            corner.x = pos.x + ((j & 4) >> 2);
            corner.y = pos.y + ((j & 2) >> 1);
            corner.z = pos.z + ((j & 1) >> 0);
            corner_idx[j] = hash_index(corner, resolution, codebook_size);
        }


        for (uint64_t j=0; j<feature_dim; ++j) {
            float gradout =  grad_output[i*num_lods*feature_dim + lod_idx*feature_dim + j];
            calc_grad_wrt_x_(j, resolution, feature_dim, x_, _x, corner_idx, gradout, grad_coords + i*3, codebook);
#           pragma unroll
            for (int k=0; k<8; ++k) {
                float grad = gradout * coeffs[k];
                atomicAdd(grad_codebook + (corner_idx[k]*feature_dim + j), grad);
            }
        }
    }
}

void hashgrid_interpolate_backward_cuda_impl(
    int64_t num_coords,
    int32_t codebook_size,
    int64_t feature_dim,
    int32_t resolution,
    int32_t lod_idx,
    int32_t num_lods,
    at::Tensor coords,
    at::Tensor grad_output,
    at::Tensor grad_codebook,
    at::Tensor codebook,
    at::Tensor grad_coords){

    int num_threads = 512;
    const at::cuda::OptionalCUDAGuard device_guard(at::device_of(grad_codebook));
    auto stream = at::cuda::getCurrentCUDAStream();
    hashgrid_interpolate_backward_cuda_kernel<<<(num_coords + num_threads - 1) / num_threads, num_threads, 0, stream>>>(
        num_coords,
        codebook_size,
        feature_dim,
        resolution,
        lod_idx,
        num_lods,
        coords.data_ptr<float>(),
        grad_output.data_ptr<float>(),
        grad_codebook.data_ptr<float>(),
        codebook.data_ptr<float>(),
        grad_coords.data_ptr<float>()
    );
}

} // namespace wisp
